#include "hip/hip_runtime.h"


#include "z.h"

#define Mask_width 5
#define Mask_radius Mask_width / 2

#define BLOCK_DIM_X 64

__global__ void Image_convolveGPUShared(float *out, float *in, int width) {

  __shared__ float sMask[Mask_width];
  __shared__ float sImage[BLOCK_DIM_X + Mask_width];

  int tidX = threadIdx.x;

  int ii = tidX + blockIdx.x * BLOCK_DIM_X;

#define P(img, x)  (((x) >= 0 && (x) < width)  ? ((img)[x]) : 0)

    sImage[tidX + Mask_radius] = P(in, ii);

    if (tidX <= Mask_radius) {
      sImage[tidX + Mask_radius] = P(in, ii - Mask_radius);
      sImage[tidX + BLOCK_DIM_X + Mask_radius] = P(in, ii + BLOCK_DIM_X);
    }

#undef P

  if (tidX < Mask_width) {
    sMask[tidX] = 1;//mask[tidX];
  }

  __syncthreads();

  if (ii < width) {
    float accum = 0;
    for (int x = -Mask_radius; x <= Mask_radius; x++) {
      float pixelValue;
      float maskValue;
      pixelValue = sImage[tidX + Mask_radius + x];
      maskValue = sMask[x + Mask_radius];
      accum += pixelValue * maskValue;
    }
    out[ii] = accum;
  }
}

int main(int argc, char *argv[]) {
  size_t dim = 1024;
  zState_t st = zState_new();
  zMemoryGroup_t in = zReadBit8Array(st, "inputVector.dat", 1, &dim);
  zMemoryGroup_t out = zMemoryGroup_new(st, zMemoryType_bit8, 1, &dim);
  zFunction_t mapFun = zFunction_new("imageConvolve", Image_convolveGPUShared);
  zMap(st, mapFun, out, in);
  zWriteBit8Array(st, "outputVector.dat", out);
  return 0;
}
